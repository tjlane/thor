#include "hip/hip_runtime.h"
/*! 
 *  Code for computing the Thompson diffraction from an atomic structure,
 *  both on the GPU and CPU.
 *
 *  First version: YTZ 2012
 *  Updated TJL 2012, 2014
 */

#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>

#include <stdexcept>
#include <iostream>
#include <fstream>
#include <sstream>

// usually bad form, but here it makes sense to include a cpp file -- this
// facilitaties compilation on GPU enabled and disabled platforms
#include "cpp_scatter.hh"
#include "cpp_scatter.cpp"


#define GBLTPB 256        // global threads per block
#define MAX_NUM_TYPES 10  // maximum number of atom types

using namespace std;

/******************************************************************************
 * GPU Only Code
 ******************************************************************************/

// ---- DEVICE CODE


// this fxn written by yutong, no longer used, retained "just in case"
// -- TJL March 2017
double __device__ atomicAdd(double* address, double val) {
     double old = *address, assumed;
     do{
         assumed = old;
         old =__longlong_as_double(atomicCAS((unsigned long long int*)address,
             __double_as_longlong(assumed),
             __double_as_longlong(val + assumed)));
     }
     while(assumed != old);
     return old;
}


template<unsigned int blockSize>
void __global__ gpu_kernel(int   const n_q,
                           float const * const __restrict__ q_x, 
                           float const * const __restrict__ q_y, 
                           float const * const __restrict__ q_z, 
             
                           int   const n_atoms,
                           float const * const __restrict__ r_x, 
                           float const * const __restrict__ r_y, 
                           float const * const __restrict__ r_z,
             
                           int   const n_atom_types,
                           int   const * const __restrict__ atom_types,
                           float const * const __restrict__ cromermann,
			   float const * const __restrict__ U,
             
                           int   const n_rotations,
                           float const * const __restrict__ q0,
                           float const * const __restrict__ q1,
                           float const * const __restrict__ q2,
                           float const * const __restrict__ q3,
             
                           float * q_out_real, // <-- not const 
                           float * q_out_imag  // <-- not const 
                          ) {
                              
    /* On-device kernel for scattering simulation
     * 
     */
    
    // int tid = threadIdx.x;
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // blank-out reduction buffer. 
    // sdata[tid] = 0;
    __syncthreads();
    
    // private variables (for each thread)
    float qx, qy, qz;             // extracted q vector
    float ax, ay, az;             // rotated r vector
    float mq, qo, fi;             // mag of q, formfactor for atom i
    float qr;                     // dot product of q and r
    float qUq;                    // matrix product of qT * U_ii * q
    
    while(gid < n_q) {
       
        // workspace for cm calcs -- static size, but hopefully big enough
        float formfactors[MAX_NUM_TYPES];
       
        // determine the rotated locations
        qx = q_x[gid];
        qy = q_y[gid];
        qz = q_z[gid];
        
        // Cromer-Mann computation, precompute for this value of q
        mq = qx*qx + qy*qy + qz*qz;
        qo = mq / (16*M_PI*M_PI); // qo is (sin(theta)/lambda)^2
        
        // accumulant: real and imaginary amplitudes for this q vector
        float2 q_sum;
        q_sum.x = 0; // x=real
        q_sum.y = 0; // y=imag

        // precompute atomic form factors for each atom type
        int tind;
        for (int type = 0; type < n_atom_types; type++) {
                
            tind = type * 9;
            fi =  cromermann[tind]   * exp(-cromermann[tind+4]*qo);
            fi += cromermann[tind+1] * exp(-cromermann[tind+5]*qo);
            fi += cromermann[tind+2] * exp(-cromermann[tind+6]*qo);
            fi += cromermann[tind+3] * exp(-cromermann[tind+7]*qo);
            fi += cromermann[tind+8];
    
            formfactors[type] = fi;

        }

        // for each molecule (2nd nested loop)
        for( int im = 0; im < n_rotations; im++ ) {
            int id;
    
            // for each atom in molecule (3rd nested loop)
            for( int a = 0; a < n_atoms; a++ ) {

                id = atom_types[a];
                fi = formfactors[id];

                rotate(r_x[a], r_y[a], r_z[a], 
                       q0[im], q1[im], q2[im], q3[im],
                       ax, ay, az);
        
                qr = ax*qx + ay*qy + az*qz;
                
		qUq_product(U, a, qx, qy, qz, qUq);

                q_sum.x += fi*__sinf(qr) * exp(- 0.5 * qUq);
                q_sum.y += fi*__cosf(qr) * exp(- 0.5 * qUq);
                
            } // finished one atom (3rd loop)
        } // finished one molecule (2nd loop)
        
        // put q 
        q_out_real[gid] = q_sum.x;
        q_out_imag[gid] = q_sum.y;

        // syncthreads are important here!
        __syncthreads();

        // offset by total working threads across all blocks. 
        gid += gridDim.x * blockDim.x;
    } // finished all pixels
}


template<unsigned int blockSize>
void __global__ gpu_diffuse_kernel(int   const n_q,
                                   float const * const __restrict__ q_x, 
                                   float const * const __restrict__ q_y, 
                                   float const * const __restrict__ q_z, 
             
                                   int   const n_atoms,
                                   float const * const __restrict__ r_x, 
                                   float const * const __restrict__ r_y, 
                                   float const * const __restrict__ r_z,
             
                                   int   const n_atom_types,
                                   int   const * const __restrict__ atom_types,
                                   float const * const __restrict__ cromermann,
             
                                   float const * const __restrict__ V,
             
                                   float * q_out_bragg,   // <-- not const 
                                   float * q_out_diffuse  // <-- not const 
                                  ) {
                              
    /* On-device kernel for scattering simulation
     * 
     */
    
    // int tid = threadIdx.x;
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    // blank-out reduction buffer. 
    // sdata[tid] = 0;
    __syncthreads();
    
    // private variables (for each thread)
    float qx, qy, qz;             // extracted q vector
    float mq, qo, fi;             // mag of q, formfactor for atom i
    float dx, dy, dz;             // difference r_i - r_j for {x,y,z}
    
    float qr;                     // dot product of q and r
    float W;                      // intermediate result
    
    float qVabq, qVaaq, qVbbq;    // matrix product qT * V_ab * q (atoms a & b)
    
    // ---> main loop (3 nested loops)
    // for each q vector (1st nested loop)
    while(gid < n_q) {
       
        // workspace for cm calcs -- static size, but hopefully big enough
        float formfactors[MAX_NUM_TYPES];
        
        // -- cache for V_ii's
        // NOT using cache for GPU... too much memory for each thread
        // to have it's own cache
        //float qViiq_cache[n_atoms];
       
        // determine the rotated locations
        qx = q_x[gid];
        qy = q_y[gid];
        qz = q_z[gid];
        
        // Cromer-Mann computation, precompute for this value of q
        mq = qx*qx + qy*qy + qz*qz;
        qo = mq / (16*M_PI*M_PI); // qo is (sin(theta)/lambda)^2
        
        // accumulant: real and imaginary amplitudes for this q vector
        float2 q_sum;
        q_sum.x = 0; // x=bragg
        q_sum.y = 0; // y=diffuse
        
        // precompute atomic form factors for each atom type
        int tind;
        for (int type = 0; type < n_atom_types; type++) {
                
            tind = type * 9;
            fi =  cromermann[tind]   * exp(-cromermann[tind+4]*qo);
            fi += cromermann[tind+1] * exp(-cromermann[tind+5]*qo);
            fi += cromermann[tind+2] * exp(-cromermann[tind+6]*qo);
            fi += cromermann[tind+3] * exp(-cromermann[tind+7]*qo);
            fi += cromermann[tind+8];
    
            formfactors[type] = fi;

        }
        
        int id_a, id_b;
        float fa, fb;

        // for each atom in molecule [again] (2nd nested loop)
        for( int a = 0; a < n_atoms; a++ ) {
            
            id_a = atom_types[a];
            fa   = formfactors[id_a];

            // do diagonal elements (a == b)
            qVq_product(V, a, a, n_atoms, qx, qy, qz, qVaaq);
            W = fa * fa;
            q_sum.x += W * exp(-1 * qVaaq);
            q_sum.y += W * (1 - exp(-1 * qVaaq));
    
            // for each atom in molecule [again], a != b (3rd nested loop)
            for( int b = 0; b < a; b++ ) {

                id_b = atom_types[b];
                fb   = formfactors[id_b];
                
                // iqr [structure factor]      
                dx = r_x[a] - r_x[b];
                dy = r_y[a] - r_y[b];
                dz = r_z[a] - r_z[b];
                qr = dx*qx + dy*qy + dz*qz;
                
                // qVq [disorder factor]
                qVq_product(V, a, b, n_atoms, qx, qy, qz, qVabq);
                qVq_product(V, b, b, n_atoms, qx, qy, qz, qVbbq);

                // accumulate (for atom pair a/b)
                W = 2 * fa * fb * cosf(qr) * exp(- 0.5 * qVaaq - 0.5 * qVbbq);
                q_sum.x += W;
                q_sum.y += W * ( exp( qVabq ) - 1 );
                
            } // finished one atom (3rd loop)
        } // finished 2nd atom (2nd loop)
        
        // put q 
        q_out_bragg[gid]   = q_sum.x;
        q_out_diffuse[gid] = q_sum.y;

        // syncthreads are important here!
        __syncthreads();

        // offset by total working threads across all blocks. 
        gid += gridDim.x * blockDim.x;
    } // finished all pixels
}


// ---- HOST CODE

void deviceMalloc( void ** ptr, int bytes ) {
    hipError_t err = hipMalloc(ptr, (size_t) bytes);
    // assert(err == 0);
}


void _gpuscatter(int device_id,
            
                 // scattering q-vectors
                 int     n_q,
                 float * h_qx,
                 float * h_qy,
                 float * h_qz,
        
                 // atomic positions, ids
                 int     n_atoms,
                 float * h_rx,
                 float * h_ry,
                 float * h_rz,

                 // cromer-mann parameters
                 int     n_atom_types,
                 int   * h_atom_types,
                 float * h_cromermann,

		 // atomic displacement parameters
		 float * h_U,

                 // random numbers for rotations
                 int     n_rotations,
                 float * rand1,
                 float * rand2,
                 float * rand3,

                 // output
                 float * h_q_out_real,
                 float * h_q_out_imag
                ) {
    
    /* This is the code to be called if nvcc is found, a GPU is around, etc
     *
     *  All arguments consist of 
     *   (1) a float pointer to the beginning of the array to be passed
     *   (2) ints representing the size of each array
     */


    // set GPU size parameters
    static const int tpb = GBLTPB;     // threads per block
    int bpg = n_q / GBLTPB + 1;        // blocks per grid
    unsigned int total_q = tpb * bpg;  // total q positions to compute
    
    
    // set the device
    hipError_t err;
    err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        printf("Error setting device ID. CUDA error: %s\n", hipGetErrorString(err));
        printf("Tried to set device to: %d\n", device_id);
        exit(-1);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Error synching device. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
    
    
    // compute the memory necessary to hold input/output
    const unsigned int q_size           = total_q * sizeof(float);
    const unsigned int r_size           = n_atoms * sizeof(float);
    const unsigned int id_size          = n_atoms * sizeof(int);
    const unsigned int cm_size          = 9 * n_atom_types * sizeof(float);
    const unsigned int quat_size        = n_rotations * sizeof(float);
    const unsigned int U_size           = n_atoms * 9 * sizeof(float);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error before device malloc. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // allocate memory on the board
    float *d_qx;         deviceMalloc( (void **) &d_qx, q_size);
    float *d_qy;         deviceMalloc( (void **) &d_qy, q_size);
    float *d_qz;         deviceMalloc( (void **) &d_qz, q_size);

    float *d_rx;         deviceMalloc( (void **) &d_rx, r_size);
    float *d_ry;         deviceMalloc( (void **) &d_ry, r_size);
    float *d_rz;         deviceMalloc( (void **) &d_rz, r_size);
    
    int   *d_id;         deviceMalloc( (void **) &d_id, id_size);
    float *d_cm;         deviceMalloc( (void **) &d_cm, cm_size);
    float *d_U;          deviceMalloc( (void **) &d_U, U_size);

    float *d_q0;         deviceMalloc( (void **) &d_q0, quat_size);
    float *d_q1;         deviceMalloc( (void **) &d_q1, quat_size);
    float *d_q2;         deviceMalloc( (void **) &d_q2, quat_size);
    float *d_q3;         deviceMalloc( (void **) &d_q3, quat_size);
    
    float *d_q_out_real; deviceMalloc( (void **) &d_q_out_real, q_size);
    float *d_q_out_imag; deviceMalloc( (void **) &d_q_out_imag, q_size);
    
    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in device malloc. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
    
    // pre-compute quaternions from random numbers
    float * h_q0 = (float *) malloc(n_rotations * sizeof(float));
    float * h_q1 = (float *) malloc(n_rotations * sizeof(float));
    float * h_q2 = (float *) malloc(n_rotations * sizeof(float));
    float * h_q3 = (float *) malloc(n_rotations * sizeof(float));
    
    for( int im = 0; im < n_rotations; im++ ) {
        generate_random_quaternion(rand1[im], rand2[im], rand3[im],
                                   h_q0[im], h_q1[im], h_q2[im], h_q3[im]);
    }

    // copy input/output arrays to board memory
    hipMemcpy(d_qx, &h_qx[0], q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &h_qy[0], q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &h_qz[0], q_size, hipMemcpyHostToDevice);
    
    hipMemcpy(d_rx, &h_rx[0], r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ry, &h_ry[0], r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rz, &h_rz[0], r_size, hipMemcpyHostToDevice);
    
    hipMemcpy(d_id, &h_atom_types[0], id_size, hipMemcpyHostToDevice);
    hipMemcpy(d_cm, &h_cromermann[0], cm_size, hipMemcpyHostToDevice);
    hipMemcpy(d_U, &h_U[0], U_size, hipMemcpyHostToDevice);    

    hipMemcpy(d_q0, &h_q0[0], quat_size, hipMemcpyHostToDevice);
    hipMemcpy(d_q1, &h_q1[0], quat_size, hipMemcpyHostToDevice);
    hipMemcpy(d_q2, &h_q2[0], quat_size, hipMemcpyHostToDevice);
    hipMemcpy(d_q3, &h_q3[0], quat_size, hipMemcpyHostToDevice);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in cuda memcpy. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // execute the kernel
    gpu_kernel<tpb> <<<bpg, tpb>>> (n_q, d_qx, d_qy, d_qz, 
                                    n_atoms, d_rx, d_ry, d_rz,
                                    n_atom_types, d_id, d_cm, d_U,
                                    n_rotations, d_q0, d_q1, d_q2, d_q3,
                                    d_q_out_real, d_q_out_imag);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in kernel. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // retrieve the output off the board and back into CPU memory
    // copys the array to the output array passed as input
    const unsigned int wanted_q_size = n_q * sizeof(float);
    hipMemcpy(&h_q_out_real[0], d_q_out_real, wanted_q_size, hipMemcpyDeviceToHost);
    hipMemcpy(&h_q_out_imag[0], d_q_out_imag, wanted_q_size, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in memcpy from device --> host. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // free memory
    hipFree(d_qx);
    hipFree(d_qy);
    hipFree(d_qz);
    
    hipFree(d_rx);
    hipFree(d_ry);
    hipFree(d_rz);
    
    hipFree(d_id);
    hipFree(d_cm);
    hipFree(d_U);
    
    hipFree(d_q0);
    hipFree(d_q1);
    hipFree(d_q2);
    hipFree(d_q3);
    
    hipFree(d_q_out_real);
    hipFree(d_q_out_imag);
    
    free(h_q0);
    free(h_q1);
    free(h_q2);
    free(h_q3);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error freeing memory. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();
    hipDeviceReset();
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error resetting device. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}




void _gpudiffuse(int device_id,
            
                 // scattering q-vectors
                 int     n_q,
                 float * h_qx,
                 float * h_qy,
                 float * h_qz,
        
                 // atomic positions, ids
                 int     n_atoms,
                 float * h_rx,
                 float * h_ry,
                 float * h_rz,

                 // cromer-mann parameters
                 int     n_atom_types,
                 int   * h_atom_types,
                 float * h_cromermann,

                 // correlation matrix
                 float * h_V,

                 // output
                 float * h_q_out_bragg,
                 float * h_q_out_diffuse
                ) {
    
    /* This is the code to be called if nvcc is found, a GPU is around, etc
     *
     *  All arguments consist of 
     *   (1) a float pointer to the beginning of the array to be passed
     *   (2) ints representing the size of each array
     */


    // set GPU size parameters
    static const int tpb = GBLTPB;     // threads per block
    int bpg = n_q / GBLTPB + 1;        // blocks per grid
    unsigned int total_q = tpb * bpg;  // total q positions to compute
    
    
    // set the device
    hipError_t err;
    err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        printf("Error setting device ID. CUDA error: %s\n", hipGetErrorString(err));
        printf("Tried to set device to: %d\n", device_id);
        exit(-1);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("Error synching device. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
    
    
    // compute the memory necessary to hold input/output
    const unsigned int q_size           = total_q * sizeof(float);
    const unsigned int r_size           = n_atoms * sizeof(float);
    const unsigned int id_size          = n_atoms * sizeof(int);
    const unsigned int cm_size          = 9 * n_atom_types * sizeof(float);
    const unsigned int V_size           = n_atoms * n_atoms * 9 * sizeof(float);


    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error before device malloc. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // allocate memory on the board
    float *d_qx;         deviceMalloc( (void **) &d_qx, q_size);
    float *d_qy;         deviceMalloc( (void **) &d_qy, q_size);
    float *d_qz;         deviceMalloc( (void **) &d_qz, q_size);

    float *d_rx;         deviceMalloc( (void **) &d_rx, r_size);
    float *d_ry;         deviceMalloc( (void **) &d_ry, r_size);
    float *d_rz;         deviceMalloc( (void **) &d_rz, r_size);
    
    int   *d_id;         deviceMalloc( (void **) &d_id, id_size);
    float *d_cm;         deviceMalloc( (void **) &d_cm, cm_size);
    
    float *d_V;          deviceMalloc( (void **) &d_V, V_size);
    float *d_q_out_bragg;   deviceMalloc( (void **) &d_q_out_bragg,   q_size);
    float *d_q_out_diffuse; deviceMalloc( (void **) &d_q_out_diffuse, q_size);
    
    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in device malloc. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // copy input/output arrays to board memory
    hipMemcpy(d_qx, &h_qx[0], q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qy, &h_qy[0], q_size, hipMemcpyHostToDevice);
    hipMemcpy(d_qz, &h_qz[0], q_size, hipMemcpyHostToDevice);
    
    hipMemcpy(d_rx, &h_rx[0], r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_ry, &h_ry[0], r_size, hipMemcpyHostToDevice);
    hipMemcpy(d_rz, &h_rz[0], r_size, hipMemcpyHostToDevice);
    
    hipMemcpy(d_id, &h_atom_types[0], id_size, hipMemcpyHostToDevice);
    hipMemcpy(d_cm, &h_cromermann[0], cm_size, hipMemcpyHostToDevice);
    
    hipMemcpy(d_V, &h_V[0], V_size, hipMemcpyHostToDevice);

    // check for errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in cuda memcpy. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // execute the kernel
    gpu_diffuse_kernel<tpb> <<<bpg, tpb>>> (n_q, d_qx, d_qy, d_qz, 
                                            n_atoms, d_rx, d_ry, d_rz,
                                            n_atom_types, d_id, d_cm, d_V,
                                            d_q_out_bragg, d_q_out_diffuse);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in kernel. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // retrieve the output off the board and back into CPU memory
    // copys the array to the output array passed as input
    const unsigned int wanted_q_size = n_q * sizeof(float);
    hipMemcpy(&h_q_out_bragg[0],   d_q_out_bragg,   wanted_q_size, hipMemcpyDeviceToHost);
    hipMemcpy(&h_q_out_diffuse[0], d_q_out_diffuse, wanted_q_size, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in memcpy from device --> host. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    // free memory
    hipFree(d_qx);
    hipFree(d_qy);
    hipFree(d_qz);
    
    hipFree(d_rx);
    hipFree(d_ry);
    hipFree(d_rz);
    
    hipFree(d_id);
    hipFree(d_cm);
    
    hipFree(d_V);
    
    hipFree(d_q_out_bragg);
    hipFree(d_q_out_diffuse);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error freeing memory. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }

    hipDeviceSynchronize();
    hipDeviceSynchronize();
    hipDeviceReset();
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error resetting device. CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

// end of GPU enabled code <---

// int main() {
//
//     int nQ_ = 1000;
//     int nAtoms_ = 1000;
//     int n_atom_types_ = 10;
//     int nRot_ = 1000;
//
//     float * h_qx_ = new float[nQ_];
//     float * h_qy_ = new float[nQ_];
//     float * h_qz_ = new float[nQ_];
//
//     float * h_rx_ = new float[nAtoms_];
//     float * h_ry_ = new float[nAtoms_];
//     float * h_rz_ = new float[nAtoms_];
//
//     int   * atom_types_ = new int[nAtoms_];
//     float * cromermann_ = new float[n_atom_types_ * 9];
//
//     float * h_rand1_ = new float[nRot_];
//     float * h_rand2_ = new float[nRot_];
//     float * h_rand3_ = new float[nRot_];
//
//     float * h_outQ_R = new float[nQ_];
//     float * h_outQ_I = new float[nQ_];
//
//     gpuscatter    ( 0, // device ID
//
//                     // q vectors
//                     nQ_,
//                     h_qx_,
//                     h_qy_,
//                     h_qz_,
//
//                     // atomic positions, ids
//                     nAtoms_,
//                     h_rx_,
//                     h_ry_,
//                     h_rz_,
//
//                     // formfactor info
//                     n_atom_types_,
//                     atom_types_,
//                     cromermann_,
//
//                     // random numbers for rotations
//                     nRot_,
//                     h_rand1_,
//                     h_rand2_,
//                     h_rand3_,
//
//                     // output
//                     h_outQ_R,
//                     h_outQ_I );
//
//     cout << h_outQ_R[0] << endl;
//     cout << h_outQ_I[0] << endl;
//
//     return 0;
// }

int main() {

    int nQ_ = 100000;
    int nAtoms_ = 1500;
    int n_atom_types_ = 10;

    float * h_qx_ = new float[nQ_];
    float * h_qy_ = new float[nQ_];
    float * h_qz_ = new float[nQ_];

    float * h_rx_ = new float[nAtoms_];
    float * h_ry_ = new float[nAtoms_];
    float * h_rz_ = new float[nAtoms_];

    int   * atom_types_ = new int[nAtoms_];
    float * cromermann_ = new float[n_atom_types_ * 9];

    float * h_V_ = new float[nAtoms_ * nAtoms_ * 3 * 3];

    float * h_outQ_R = new float[nQ_];
    float * h_outQ_I = new float[nQ_];

    gpudiffuse    ( 0, // device ID

                    // q vectors
                    nQ_,
                    h_qx_,
                    h_qy_,
                    h_qz_,

                    // atomic positions, ids
                    nAtoms_,
                    h_rx_,
                    h_ry_,
                    h_rz_,

                    // formfactor info
                    n_atom_types_,
                    atom_types_,
                    cromermann_,

                    // random numbers for rotations
                    h_V_,

                    // output
                    h_outQ_R,
                    h_outQ_I );

    cout << h_outQ_R[0] << endl;
    cout << h_outQ_I[0] << endl;

    return 0;
}
